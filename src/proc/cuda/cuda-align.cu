
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-align.cuh"
#include "../../../include/librealsense2/rsutil.h"
#include "../../cuda/rscuda_utils.cuh"

// CUDA headers
#include <cuda_runtime.h>

#ifdef _MSC_VER 
// Add library dependencies if using VS
#pragma comment(lib, "cudart_static")
#endif

#define RS2_CUDA_THREADS_PER_BLOCK 32

using namespace librealsense;
using namespace rscuda;

template<int N> struct bytes { unsigned char b[N]; };

int calc_block_size(int pixel_count, int thread_count)
{
    return ((pixel_count % thread_count) == 0) ? (pixel_count / thread_count) : (pixel_count / thread_count + 1);
}

__device__ void kernel_transfer_pixels(int2* mapped_pixels, const rs2_intrinsics* depth_intrin,
    const rs2_intrinsics* other_intrin, const rs2_extrinsics* depth_to_other, float depth_val, int depth_x, int depth_y, int block_index)
{
    float shift = block_index ? 0.5 : -0.5;
    auto depth_size = depth_intrin->width * depth_intrin->height;
    auto mapped_index = block_index * depth_size + (depth_y * depth_intrin->width + depth_x);

    if (mapped_index >= depth_size * 2)
        return;

    // Skip over depth pixels with the value of zero, we have no depth data so we will not write anything into our aligned images
    if (depth_val == 0)
    {
        mapped_pixels[mapped_index] = { -1, -1 };
        return;
    }

    //// Map the top-left corner of the depth pixel onto the other image
    float depth_pixel[2] = { depth_x + shift, depth_y + shift }, depth_point[3], other_point[3], other_pixel[2];
    rscuda::rs2_deproject_pixel_to_point(depth_point, depth_intrin, depth_pixel, depth_val);
    rscuda::rs2_transform_point_to_point(other_point, depth_to_other, depth_point);
    rscuda::rs2_project_point_to_pixel(other_pixel, other_intrin, other_point);
    mapped_pixels[mapped_index].x = static_cast<int>(other_pixel[0] + 0.5f);
    mapped_pixels[mapped_index].y = static_cast<int>(other_pixel[1] + 0.5f);
}

__global__  void kernel_map_depth_to_other(int2* mapped_pixels, const uint16_t* depth_in, const rs2_intrinsics* depth_intrin, const rs2_intrinsics* other_intrin,
    const rs2_extrinsics* depth_to_other, float depth_scale)
{
    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

    int depth_pixel_index = depth_y * depth_intrin->width + depth_x;
    if (depth_pixel_index >= depth_intrin->width * depth_intrin->height)
        return;
    float depth_val = depth_in[depth_pixel_index] * depth_scale;
    kernel_transfer_pixels(mapped_pixels, depth_intrin, other_intrin, depth_to_other, depth_val, depth_x, depth_y, blockIdx.z);
}

template<int BPP>
__global__  void kernel_other_to_depth(unsigned char* aligned, const unsigned char* other, const int2* mapped_pixels, const rs2_intrinsics* depth_intrin, const rs2_intrinsics* other_intrin)
{
    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

    auto depth_size = depth_intrin->width * depth_intrin->height;
    int depth_pixel_index = depth_y * depth_intrin->width + depth_x;

    if (depth_pixel_index >= depth_intrin->width * depth_intrin->height)
        return;

    int2 p0 = mapped_pixels[depth_pixel_index];
    int2 p1 = mapped_pixels[depth_size + depth_pixel_index];

    if (p0.x < 0 || p0.y < 0 || p1.x >= other_intrin->width || p1.y >= other_intrin->height)
        return;

    // Transfer between the depth pixels and the pixels inside the rectangle on the other image
    auto in_other = (const bytes<BPP> *)(other);
    auto out_other = (bytes<BPP> *)(aligned);
    for (int y = p0.y; y <= p1.y; ++y)
    {
        for (int x = p0.x; x <= p1.x; ++x)
        {
            auto other_pixel_index = y * other_intrin->width + x;
            out_other[depth_pixel_index] = in_other[other_pixel_index];
        }
    }
}

__global__  void kernel_depth_to_other(uint16_t* aligned_out, const uint16_t* depth_in, const int2* mapped_pixels, const rs2_intrinsics* depth_intrin, const rs2_intrinsics* other_intrin)
{
    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

    auto depth_size = depth_intrin->width * depth_intrin->height;
    int depth_pixel_index = depth_y * depth_intrin->width + depth_x;

    if (depth_pixel_index >= depth_intrin->width * depth_intrin->height)
        return;

    int2 p0 = mapped_pixels[depth_pixel_index];
    int2 p1 = mapped_pixels[depth_size + depth_pixel_index];

    if (p0.x < 0 || p0.y < 0 || p1.x >= other_intrin->width || p1.y >= other_intrin->height)
        return;

    // Transfer between the depth pixels and the pixels inside the rectangle on the other image
    unsigned int new_val = depth_in[depth_pixel_index];
    unsigned int* arr = (unsigned int*)aligned_out;
    for (int y = p0.y; y <= p1.y; ++y)
    {
        for (int x = p0.x; x <= p1.x; ++x)
        {
            auto other_pixel_index = y * other_intrin->width + x;
            new_val = new_val << 16 | new_val;
            atomicMin(&arr[other_pixel_index / 2], new_val);
        }
    }
}

__global__  void kernel_replace_to_zero(uint16_t* aligned_out, const rs2_intrinsics* other_intrin)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    auto other_pixel_index = y * other_intrin->width + x;
    if (aligned_out[other_pixel_index] == 0xffff)
        aligned_out[other_pixel_index] = 0;
}

void align_cuda_helper::align_other_to_depth(unsigned char* h_aligned_out, const uint16_t* h_depth_in,
    float depth_scale, const rs2_intrinsics& h_depth_intrin, const rs2_extrinsics& h_depth_to_other,
    const rs2_intrinsics& h_other_intrin, const unsigned char* h_other_in, rs2_format other_format, int other_bytes_per_pixel)
{
    int depth_pixel_count = h_depth_intrin.width * h_depth_intrin.height;
    int other_pixel_count = h_other_intrin.width * h_other_intrin.height;
    int depth_size = depth_pixel_count * 2;
    int other_size = other_pixel_count * other_bytes_per_pixel;
    int aligned_pixel_count = depth_pixel_count;
    int aligned_size = aligned_pixel_count * other_bytes_per_pixel;

    // allocate and copy objects to cuda device memory
    if (!_d_depth_intrinsics) _d_depth_intrinsics = make_device_copy(h_depth_intrin);
    if (!_d_other_intrinsics) _d_other_intrinsics = make_device_copy(h_other_intrin);
    if (!_d_depth_other_extrinsics) _d_depth_other_extrinsics = make_device_copy(h_depth_to_other);

    if (!_d_depth_in) _d_depth_in = alloc_dev<uint16_t>(aligned_pixel_count);
    cudaMemcpy(_d_depth_in.get(), h_depth_in, depth_size, cudaMemcpyHostToDevice);

    if (!_d_other_in) _d_other_in = alloc_dev<unsigned char>(other_size);
    cudaMemcpy(_d_other_in.get(), h_other_in, other_size, cudaMemcpyHostToDevice);

    if (!_d_aligned_out)
        _d_aligned_out = alloc_dev<unsigned char>(aligned_size);
    cudaMemset(_d_aligned_out.get(), 0, aligned_size);

    if (!_d_pixel_map) _d_pixel_map = alloc_dev<int2>(depth_pixel_count * 2);

    // config threads
    dim3 threads(RS2_CUDA_THREADS_PER_BLOCK, RS2_CUDA_THREADS_PER_BLOCK);
    dim3 depth_blocks(calc_block_size(h_depth_intrin.width, threads.x), calc_block_size(h_depth_intrin.height, threads.y));
    dim3 mapping_blocks(depth_blocks.x, depth_blocks.y, 2);

    kernel_map_depth_to_other <<<mapping_blocks,threads>>> (_d_pixel_map.get(), _d_depth_in.get(), _d_depth_intrinsics.get(), _d_other_intrinsics.get(),
        _d_depth_other_extrinsics.get(), depth_scale);

    switch (other_bytes_per_pixel)
    {
    case 1: kernel_other_to_depth<1> <<<depth_blocks,threads>>> (_d_aligned_out.get(), _d_other_in.get(), _d_pixel_map.get(), _d_depth_intrinsics.get(), _d_other_intrinsics.get()); break;
    case 2: kernel_other_to_depth<2> <<<depth_blocks,threads>>> (_d_aligned_out.get(), _d_other_in.get(), _d_pixel_map.get(), _d_depth_intrinsics.get(), _d_other_intrinsics.get()); break;
    case 3: kernel_other_to_depth<3> <<<depth_blocks,threads>>> (_d_aligned_out.get(), _d_other_in.get(), _d_pixel_map.get(), _d_depth_intrinsics.get(), _d_other_intrinsics.get()); break;
    case 4: kernel_other_to_depth<4> <<<depth_blocks,threads>>> (_d_aligned_out.get(), _d_other_in.get(), _d_pixel_map.get(), _d_depth_intrinsics.get(), _d_other_intrinsics.get()); break;
    }

    cudaDeviceSynchronize();

    cudaMemcpy(h_aligned_out, _d_aligned_out.get(), aligned_size, cudaMemcpyDeviceToHost);
}

void align_cuda_helper::align_depth_to_other(unsigned char* h_aligned_out, const uint16_t* h_depth_in,
    float depth_scale, const rs2_intrinsics& h_depth_intrin, const rs2_extrinsics& h_depth_to_other,
    const rs2_intrinsics& h_other_intrin)
{
    int depth_pixel_count = h_depth_intrin.width * h_depth_intrin.height;
    int other_pixel_count = h_other_intrin.width * h_other_intrin.height;
    int aligned_pixel_count = other_pixel_count;

    int depth_byte_size = depth_pixel_count * 2;
    int aligned_byte_size = aligned_pixel_count * 2;

    // allocate and copy objects to cuda device memory
    if (!_d_depth_intrinsics) _d_depth_intrinsics = make_device_copy(h_depth_intrin);
    if (!_d_other_intrinsics) _d_other_intrinsics = make_device_copy(h_other_intrin);
    if (!_d_depth_other_extrinsics) _d_depth_other_extrinsics = make_device_copy(h_depth_to_other);

    if (!_d_depth_in) _d_depth_in = alloc_dev<uint16_t>(depth_pixel_count);
    cudaMemcpy(_d_depth_in.get(), h_depth_in, depth_byte_size, cudaMemcpyHostToDevice);

    if (!_d_aligned_out) _d_aligned_out = alloc_dev<unsigned char>(aligned_byte_size);
    cudaMemset(_d_aligned_out.get(), 0xff, aligned_byte_size);

    if (!_d_pixel_map) _d_pixel_map = alloc_dev<int2>(depth_pixel_count * 2);

    // config threads
    dim3 threads(RS2_CUDA_THREADS_PER_BLOCK, RS2_CUDA_THREADS_PER_BLOCK);
    dim3 depth_blocks(calc_block_size(h_depth_intrin.width, threads.x), calc_block_size(h_depth_intrin.height, threads.y));
    dim3 other_blocks(calc_block_size(h_other_intrin.width, threads.x), calc_block_size(h_other_intrin.height, threads.y));
    dim3 mapping_blocks(depth_blocks.x, depth_blocks.y, 2);

    kernel_map_depth_to_other <<<mapping_blocks,threads>>> (_d_pixel_map.get(), _d_depth_in.get(), _d_depth_intrinsics.get(),
        _d_other_intrinsics.get(), _d_depth_other_extrinsics.get(), depth_scale);

    kernel_depth_to_other <<<depth_blocks,threads>>> ((uint16_t*)_d_aligned_out.get(), _d_depth_in.get(), _d_pixel_map.get(),
        _d_depth_intrinsics.get(), _d_other_intrinsics.get());

    kernel_replace_to_zero <<<other_blocks, threads>>> ((uint16_t*)_d_aligned_out.get(), _d_other_intrinsics.get());

    cudaDeviceSynchronize();

    cudaMemcpy(h_aligned_out, _d_aligned_out.get(), aligned_pixel_count * 2, cudaMemcpyDeviceToHost);
}

#endif //RS2_USE_CUDA
