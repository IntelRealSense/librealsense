
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-pointcloud.cuh"
#include <iostream>
#include <chrono>


__device__
float map_depth (float depth_scale, uint16_t z) {
    return depth_scale * z;
}

__device__
void deproject_pixel_to_point_cuda(float points[3], const struct rs2_intrinsics * intrin, const float pixel[2], float depth) {
    assert(intrin->model != RS2_DISTORTION_MODIFIED_BROWN_CONRADY); // Cannot deproject from a forward-distorted image
    assert(intrin->model != RS2_DISTORTION_FTHETA); // Cannot deproject to an ftheta image
    //assert(intrin->model != RS2_DISTORTION_BROWN_CONRADY); // Cannot deproject to an brown conrady model
    float x = (pixel[0] - intrin->ppx) / intrin->fx;
    float y = (pixel[1] - intrin->ppy) / intrin->fy;    

    float xo = x;
    float yo = y;

    if (intrin->model == RS2_DISTORTION_INVERSE_BROWN_CONRADY)
    {
        // need to loop until convergence 
        // 10 iterations determined empirically
        for (int i = 0; i < 10; i++)
        {
            float r2 = x * x + y * y;
            float icdist = (float)1 / (float)(1 + ((intrin->coeffs[4] * r2 + intrin->coeffs[1])*r2 + intrin->coeffs[0])*r2);
            float xq = x / icdist;
            float yq = y / icdist;
            float delta_x = 2 * intrin->coeffs[2] * xq*yq + intrin->coeffs[3] * (r2 + 2 * xq*xq);
            float delta_y = 2 * intrin->coeffs[3] * xq*yq + intrin->coeffs[2] * (r2 + 2 * yq*yq);
            x = (xo - delta_x)*icdist;
            y = (yo - delta_y)*icdist;
        }
    }
    else if (intrin->model == RS2_DISTORTION_BROWN_CONRADY)
    {
        // need to loop until convergence 
        // 10 iterations determined empirically
        for (int i = 0; i < 10; i++)
        {
            float r2 = x * x + y * y;
            float icdist = (float)1 / (float)(1 + ((intrin->coeffs[4] * r2 + intrin->coeffs[1])*r2 + intrin->coeffs[0])*r2);
            float delta_x = 2 * intrin->coeffs[2] * x*y + intrin->coeffs[3] * (r2 + 2 * x*x);
            float delta_y = 2 * intrin->coeffs[3] * x*y + intrin->coeffs[2] * (r2 + 2 * y*y);
            x = (xo - delta_x)*icdist;
            y = (yo - delta_y)*icdist;
        }
    }
    points[0] = depth * x;
    points[1] = depth * y;
    points[2] = depth;
    
}


__global__
//void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, std::function<uint16_t(float)> map_depth)

void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics* intrin, const uint16_t * depth, float depth_scale)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i >= (*intrin).height * (*intrin).width) {
        return;
    }
    int stride = blockDim.x * gridDim.x;
    int a, b;
    
    for (int j = i; j < (*intrin).height * (*intrin).width; j += stride) {
        b = j / (*intrin).width;
        a = j - b * (*intrin).width;
        const float pixel[] = { (float)a, (float)b };
        deproject_pixel_to_point_cuda(points + j * 3, intrin, pixel, depth_scale * depth[j]);               
   }
}


void rscuda::deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, float depth_scale)
{
    int count = intrin.height * intrin.width;
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    float *dev_points = 0;	
    uint16_t *dev_depth = 0;
    rs2_intrinsics* dev_intrin = 0;
    cudaError_t result;

    result = cudaMalloc(&dev_points, count * sizeof(float) * 3);
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_depth, count * sizeof(uint16_t));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_intrin, sizeof(rs2_intrinsics));
    assert(result == cudaSuccess);
       
    result = cudaMemcpy(dev_depth, depth, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess); 
    result = cudaMemcpy(dev_intrin, &intrin, sizeof(rs2_intrinsics), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess); 
     
    kernel_deproject_depth_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(dev_points, dev_intrin, dev_depth, depth_scale); 

     result = cudaMemcpy(points, dev_points, count * sizeof(float) * 3, cudaMemcpyDeviceToHost);
     assert(result == cudaSuccess);

    cudaFree(dev_points);
    cudaFree(dev_depth);
    cudaFree(dev_intrin);
}

#endif
