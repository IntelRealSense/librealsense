
#include <hip/hip_runtime.h>
//This file is partly based on Mehran Maghoumi's work: https://github.com/Maghoumi/culibrealsense

#ifdef RS2_USE_CUDA

#include "cuda-conversion.cuh"
#include <iostream>
#include <iomanip>
#include "rscuda_utils.cuh"
/*
// conversion to Y8 is currently not available in the API
__global__ void kernel_unpack_yuy2_y8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= superPixCount)
        return;

    int idx = i * 4;

    dst[idx] = src[idx];
    dst[idx + 1] = src[idx + 2];
    dst[idx + 2] = src[idx + 4];
    dst[idx + 3] = src[idx + 6];
    dst[idx + 4] = src[idx + 8];
    dst[idx + 5] = src[idx + 10];
    dst[idx + 6] = src[idx + 12];
    dst[idx + 7] = src[idx + 14];
    dst[idx + 8] = src[idx + 16];
    dst[idx + 9] = src[idx + 18];
    dst[idx + 10] = src[idx + 20];
    dst[idx + 11] = src[idx + 22];
    dst[idx + 12] = src[idx + 24];
    dst[idx + 13] = src[idx + 26];
    dst[idx + 14] = src[idx + 28];
    dst[idx + 15] = src[idx + 30];
}
*/

__global__ void kernel_unpack_yuy2_y16_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i >= superPixCount)
        return;

    for (; i < superPixCount; i += stride) {

        int idx = i * 4;

        dst[idx] = 0;
        dst[idx + 1] = src[idx + 0];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];
    }
}


__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i >= superPixCount)
        return;

    for (; i < superPixCount; i += stride) {

        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 6;

        dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);

        c = y1 - 16;

        dst[odx + 3] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 5] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp

    }
}

__global__ void kernel_unpack_yuy2_bgr8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i >= superPixCount)
        return;

    for (; i < superPixCount; i += stride) {

        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 6;

        dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx] = clamp((298 * c + 516 * d + 128) >> 8);

        c = y1 - 16;

        dst[odx + 5] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 3] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp
    }
}


__global__ void kernel_unpack_yuy2_rgba8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i >= superPixCount)
        return;

    for (; i < superPixCount; i += stride) {

        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 8;

        dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);
        dst[odx + 3] = 255;

        c = y1 - 16;

        dst[odx + 4] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 6] = clamp((298 * c + 516 * d + 128) >> 8);
        dst[odx + 7] = 255;

#undef clamp
    }
}

__global__ void kernel_unpack_yuy2_bgra8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i >= superPixCount)
        return;

    for (; i < superPixCount; i += stride) {

        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int32_t t;

#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 8;

        dst[odx + 3] = 255;
        dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx] = clamp((298 * c + 516 * d + 128) >> 8);

        c = y1 - 16;

        dst[odx + 7] = 255;
        dst[odx + 6] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp
    }
}


void rscuda::unpack_yuy2_cuda_helper(const uint8_t* h_src, uint8_t* h_dst, int n, rs2_format format)
{
    /*    cudaEvent_t start, stop;
        cudaEventCreate(&start);
        cudaEventCreate(&stop);
        cudaEventRecord(start); */

        // How many super pixels do we have?
    int superPix = n / 2;
    std::shared_ptr<uint8_t> d_dst;
    std::shared_ptr<uint8_t> d_src = alloc_dev<uint8_t>(superPix * 4);

    auto result = cudaMemcpy(d_src.get(), h_src, superPix * sizeof(uint8_t) * 4, cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    int numBlocks = superPix / RS2_CUDA_THREADS_PER_BLOCK;
    int size;

    switch (format)
    {
        // conversion to Y8 is currently not available in the API
        /*	case RS2_FORMAT_Y8:
            size = 1;
             d_dst = alloc_dev<uint8_t>(n * size);
            kernel_unpack_yuy2_y8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
            break;
        */
    case RS2_FORMAT_Y16:
        size = 2;
        d_dst = alloc_dev<uint8_t>(n * size);
        kernel_unpack_yuy2_y16_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_src.get(), d_dst.get(), superPix);
        break;
    case RS2_FORMAT_RGB8:
        size = 3;
        d_dst = alloc_dev<uint8_t>(n * size);
        kernel_unpack_yuy2_rgb8_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_src.get(), d_dst.get(), superPix);
        break;
    case RS2_FORMAT_BGR8:
        size = 3;
        d_dst = alloc_dev<uint8_t>(n * size);
        kernel_unpack_yuy2_bgr8_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_src.get(), d_dst.get(), superPix);
        break;
    case RS2_FORMAT_RGBA8:
        size = 4;
        d_dst = alloc_dev<uint8_t>(n * size);
        kernel_unpack_yuy2_rgba8_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_src.get(), d_dst.get(), superPix);
        break;
    case RS2_FORMAT_BGRA8:
        size = 4;
        d_dst = alloc_dev<uint8_t>(n * size);
        kernel_unpack_yuy2_bgra8_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_src.get(), d_dst.get(), superPix);
        break;
    default:
        assert(false);
    }
    result = cudaGetLastError();
    assert(result == cudaSuccess);

    cudaDeviceSynchronize();

    result = cudaMemcpy(h_dst, d_dst.get(), n * sizeof(uint8_t) * size, cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    /*	cudaEventRecord(stop);
        cudaEventSynchronize(stop);
        float milliseconds = 0;
        cudaEventElapsedTime(&milliseconds, start, stop);
        std::cout << milliseconds << "\n"; */
}


__global__ void kernel_split_frame_y8_y8_from_y8i_cuda(uint8_t* a, uint8_t* b, int count, const rscuda::y8i_pixel * source)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= count)
        return;

    a[i] = source[i].l;
    b[i] = source[i].r;
}

void rscuda::y8_y8_from_y8i_cuda_helper(uint8_t* const dest[], int count, const rscuda::y8i_pixel * source)
{
    /*    cudaEvent_t start, stop;
        cudaEventCreate(&start);
        cudaEventCreate(&stop);
        cudaEventRecord(start); */

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    uint8_t* a = dest[0];
    uint8_t* b = dest[1];

    auto d_src = alloc_dev<rscuda::y8i_pixel>(count);
    auto d_dst_0 = alloc_dev<uint8_t>(count);
    auto d_dst_1 = alloc_dev<uint8_t>(count);

    auto result = cudaMemcpy(d_src.get(), source, count * sizeof(rscuda::y8i_pixel), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    kernel_split_frame_y8_y8_from_y8i_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_dst_0.get(), d_dst_1.get(), count, d_src.get());
    cudaDeviceSynchronize();

    result = cudaGetLastError();
    assert(result == cudaSuccess);

    result = cudaMemcpy(a, d_dst_0.get(), count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    result = cudaMemcpy(b, d_dst_1.get(), count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    /*    cudaEventRecord(stop);
        cudaEventSynchronize(stop);
        float milliseconds = 0;
        cudaEventElapsedTime(&milliseconds, start, stop);
        std::cout << milliseconds << std::endl; */
}

__global__ void kernel_split_frame_y16_y16_from_y12i_cuda(uint16_t* a, uint16_t* b, int count, const rscuda::y12i_pixel * source)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= count)
        return;

    a[i] = source[i].l() << 6 | source[i].l() >> 4;
    b[i] = source[i].r() << 6 | source[i].r() >> 4;
}


void rscuda::y16_y16_from_y12i_10_cuda_helper(uint8_t* const dest[], int count, const rscuda::y12i_pixel * source)
{
    /*
        cudaEvent_t start, stop;
        cudaEventCreate(&start);
        cudaEventCreate(&stop);
        cudaEventRecord(start); */

    source = reinterpret_cast<const y12i_pixel*>(source);

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    uint16_t* a = reinterpret_cast<uint16_t*>(dest[0]);
    uint16_t* b = reinterpret_cast<uint16_t*>(dest[1]);

    auto d_src = alloc_dev<rscuda::y12i_pixel>(count);
    auto d_dst_0 = alloc_dev<uint16_t>(count);
    auto d_dst_1 = alloc_dev<uint16_t>(count);


    auto result = cudaMemcpy(d_src.get(), source, count * sizeof(rscuda::y12i_pixel), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    kernel_split_frame_y16_y16_from_y12i_cuda <<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>> (d_dst_0.get(), d_dst_1.get(), count, d_src.get());
    cudaDeviceSynchronize();

    result = cudaGetLastError();
    assert(result == cudaSuccess);

    result = cudaMemcpy(a, d_dst_0.get(), count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    result = cudaMemcpy(b, d_dst_1.get(), count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    /*
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << std::endl;
    */
}


__global__ void kernel_z16_y8_from_sr300_inzi_cuda(const uint16_t* source, uint8_t* const dest, int count)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= count)
        return;

    dest[i] = source[i] >> 2;
}

void rscuda::unpack_z16_y8_from_sr300_inzi_cuda(uint8_t * const dest, const uint16_t * source, int count)
{
    /*  cudaEvent_t start, stop;
        cudaEventCreate(&start);
        cudaEventCreate(&stop);
        cudaEventRecord(start); */

    auto d_src = alloc_dev<uint16_t>(count);
    auto d_dst = alloc_dev<uint8_t>(count);

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;

    auto result = cudaMemcpy(d_src.get(), source, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    kernel_z16_y8_from_sr300_inzi_cuda <<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>> (d_src.get(), d_dst.get(), count);
    cudaDeviceSynchronize();

    result = cudaMemcpy(dest, d_dst.get(), count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    /*  cudaEventRecord(stop);
        cudaEventSynchronize(stop);
        float milliseconds = 0;
        cudaEventElapsedTime(&milliseconds, start, stop);
        std::cout << milliseconds << std::endl; */
}

__global__ void kernel_z16_y16_from_sr300_inzi_cuda(uint16_t* const source, uint16_t* const dest, int count)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= count)
        return;

    dest[i] = source[i] << 6;
}

void rscuda::unpack_z16_y16_from_sr300_inzi_cuda(uint16_t * const dest, const uint16_t * source, int count)
{
    /*  cudaEvent_t start, stop;
        cudaEventCreate(&start);
        cudaEventCreate(&stop);
        cudaEventRecord(start); */

    auto d_src = alloc_dev<uint16_t>(count);
    auto d_dst = alloc_dev<uint16_t>(count);

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;

    auto result = cudaMemcpy(d_src.get(), source, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    kernel_z16_y16_from_sr300_inzi_cuda << <numBlocks, RS2_CUDA_THREADS_PER_BLOCK >> > (d_src.get(), d_dst.get(), count);
    cudaDeviceSynchronize();

    result = cudaMemcpy(dest, d_dst.get(), count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    /*	cudaEventRecord(stop);
        cudaEventSynchronize(stop);
        float milliseconds = 0;
        cudaEventElapsedTime(&milliseconds, start, stop);
        std::cout << milliseconds << std::endl; */
}

#endif
